#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCK SIZE 16

__global void Muld(float* A float* B int wA int wB float* C) { global__ void Muld(float* A, float* B, int wA, int wB, float* C) {
  int bx = blockIdx.x; // Block index
  int by = blockIdx.y;
  i t t th dId int tx = threadIdx.x; // Th d i d // Thread index
  int ty = threadIdx.y;
  int aBegin = wA * BLOCK_SIZE * by; // Index of the first sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1; // Index of the last sub-matrix of A processed by the block
  int aStep = BLOCK_SIZE; // Step size used to iterate through the sub-matrices of A
  int bBegin = BLOCK_SIZE * bx; // Index of the first sub-matrix of B processed by the block
  int bStep = BLOCK_SIZE * wB; // Step size used to iterate through the sub-matrices of B
  float Csub = 0; // The element of the block sub-matrix that is computed by the thread
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
      // Shared memory for the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
// Shared memory for the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    As[ty][tx] = A[a + wA * ty + tx]; // Load the matrices from global memory to shared memory;
    Bs[ty][tx] = B[b + wB * ty + tx]; // each thread loads one element of each matrix
    __syncthreads(); // Synchronize to make sure the matrices are loaded
// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
      for (int k = 0; k < BLOCK_SIZE; ++k)
        Csub += As[ty][k] * Bs[k][tx];
// Synchronize to make sure that the preceding
// computation is done before loading two new
// sub-matrices of A and B in the next iteration
    __syncthreads();
  }
// Write the block sub-matrix to global memory;
// each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

int main(const float* A, const float* B, int hA, int wA, int wB, float* C) {
  int size;
  // L d A d B t th d i // Load A and B to the device
  float* Ad; size = hA * wA * sizeof(float); hipMalloc((void**)&Ad, size);
  hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
  float* Bd; size = wA * wB * sizeof(float); hipMalloc((void**)&Bd, size);
  hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
// Allocate C on the device
  float* Cd;
  size = hA * wB * sizeof(float);
  hipMalloc((void**)&Cd, size);
  // Compute the execution configuration assuming the matrix dimensions are multiples of BLOCK_SIZE
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(wB / dimBlock.x, hA / dimBlock.y);
  // Launch the device computation
  Muld<<<dimGrid, dimBlock>>>(Ad, Bd, wA, wB, Cd);
  // Read C from the device
  hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(Ad);
  hipFree(Bd);
  hipFree(Cd);

  return 0;
}
