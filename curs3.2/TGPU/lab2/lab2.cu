#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if(_m_cudaStat != hipSuccess) {\
    fprintf(stderr, "Error %s at line %d in file %s\n",\
    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
    exit(1);\
    }}

double wtime() {
    struct timeval t;
    gettimeofday(&t, NULL);
    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

__global__ void gpuSum(float *a, float *b, float *c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
    
//    c[i] *= c[i];

}

int main() {
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, t;
    int threads_per_block = 1;
    int N = pow(2,23);
    int num_of_blocks = N / threads_per_block;
    
    h_a = (float*)calloc(N, sizeof(float));
    h_b = (float*)calloc(N, sizeof(float));
    h_c = (float*)calloc(N, sizeof(float));
    
    for(int i = 0; i < N; i++) {
	h_a[i] = i * 2;
	h_b[i] = i * 3; 
    }
    
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_a, N*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_b, N*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_c, N*sizeof(float)));

    CUDA_CHECK_RETURN(hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice));
    
    //double t = wtime();
    hipEventRecord(start,0);
    gpuSum<<<dim3(num_of_blocks),dim3(threads_per_block)>>>(d_a, d_b, d_c);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    //CUDA_CHECK_RETURN(cudaDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    //t = wtime() - t;
    hipEventElapsedTime(&t, start, stop);
    
    CUDA_CHECK_RETURN(hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost));
    
//    for(int i = 0; i < N; i++) {
//	printf("%g\n", h_c[i]);
//    }
    
    fprintf(stderr, "Elapsed Time %g\n", t);
    //printf("Elapsed time: %.6fsec. \n", t);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}
