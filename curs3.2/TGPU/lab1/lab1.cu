#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

double wtime() {
    struct timeval t;
    gettimeofday(&t, NULL);
    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

__global__ void gpuSum(float *a, float *b, float *c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
    
//    c[i] *= c[i];

}

int main() {
    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
    int threads_per_block = 1024;
    int N = pow(2,10);
    int num_of_blocks = N / threads_per_block;
    
    h_a = (float*)calloc(N, sizeof(float));
    h_b = (float*)calloc(N, sizeof(float));
    h_c = (float*)calloc(N, sizeof(float));
    
    for(int i = 0; i < N; i++) {
	h_a[i] = i * 2;
	h_b[i] = i * 3; 
    }
    hipMalloc((void**)&d_a, N*sizeof(float));
    hipMalloc((void**)&d_b, N*sizeof(float));
    hipMalloc((void**)&d_c, N*sizeof(float));

    hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);
    
    double t = wtime();
    gpuSum<<<dim3(num_of_blocks),dim3(threads_per_block)>>>(d_a, d_b, d_c);
    
    hipDeviceSynchronize();
    t = wtime() - t;
    
    hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);
    
//    for(int i = 0; i < N; i++) {
//	printf("%g\n", h_c[i]);
//    }
    
    printf("Elapsed time: %.6fsec. \n", t);
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}
